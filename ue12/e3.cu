#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    __shared__ float  sub[16][16];

    // now calculate the value at that position
    const float period = 128.0f;

    sub[threadIdx.x][threadIdx.y] =
            255 * (sinf(x*2.0f*PI/ period) + 1.0f) *
                  (sinf(y*2.0f*PI/ period) + 1.0f) / 4.0f;
    __syncthreads();
	
	//assign color for each pixel position
    ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = sub[15-threadIdx.x][15-threadIdx.y];
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {

	//name a bitmap and assign values
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

    hipMalloc( (void**)&dev_bitmap, bitmap.image_size());
    data.dev_bitmap = dev_bitmap;

    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( dev_bitmap );

    hipMemcpy( bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost );
                              
    hipFree( dev_bitmap );
                              
    bitmap.display_and_exit();
}
