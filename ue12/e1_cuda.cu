#include <stdio.h>
#include "hip/hip_runtime.h"

#define DIM 20
#define TURNS 1


__global__ void gridmean(float grid[DIM][DIM], float tmp_grid[DIM][DIM]){
	int x = blockIdx.x * blockDim.x;
	int y = blockIdx.y * blockDim.y;

	float tmp = 0;
	for(int i = x-1; i <= x+1; i++){
		for(int j = y-1; j <= y+1; j++){
			if(!(i < 0 || j < 0|| i >= DIM || j >= DIM)){
				tmp += grid[i][j];
			}	
		}
	}
	tmp_grid[x][y] = tmp/9;

}

__global__ void copy(float grid[DIM][DIM], float tmp_grid[DIM][DIM]){
	grid[blockIdx.x][blockIdx.y] = tmp_grid[blockIdx.x][blockIdx.y];
}

int main(){
	
	float grid[DIM][DIM];
	float tmp_grid[DIM][DIM];
	float (*grid_d)[DIM];
	float (*tmp_grid_d)[DIM];
	int size = DIM*DIM*sizeof(float);
	hipMalloc((void**)&grid_d,size);
	hipMalloc((void**)&tmp_grid_d,size);

	for(int i = 0; i < DIM; i++){
		for(int j = 0; j < DIM; j++){
			grid[i][j] = (float) i*DIM + j;
		}
	}
	
	//for(int i = 0; i < DIM; i++){
	//	for(int j = 0; j < DIM; j++){
	//		printf("%f ", grid[i][j]);
	//	}
	//	printf("\n");
	//}

	hipMemcpy(grid_d,grid,size, hipMemcpyHostToDevice);
	hipMemcpy(tmp_grid_d,tmp_grid,size, hipMemcpyHostToDevice);
	dim3 numBlocks(DIM,DIM);

	for(int k = 0; k < TURNS; k++){
		gridmean<<<numBlocks,1>>>(grid_d,tmp_grid_d);
		copy<<<numBlocks,1>>>(grid_d,tmp_grid_d);
	}

	hipMemcpy(grid,grid_d,size, hipMemcpyDeviceToHost);

	for(int i = 0; i < DIM; i++){
		for(int j = 0; j < DIM; j++){
			printf("%f ", grid[i][j]);
		}
		printf("\n");
	}
	hipFree(grid_d);
	return 0;
}
