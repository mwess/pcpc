
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define DIM 100

using namespace std;

struct point {
	float   r;
	float   i;
	//construct function
	__device__ point( float a, float b )  {r=a; i=b;}

	__device__ float magnitude2() { return r * r + i * i; }
	
	//override the operator *
    	__device__ point operator*(const point& a) {
        	return point(r*a.r - i*a.i, i*a.r + r*a.i);
    	}

	//override the operator +
    	__device__ point operator+(const point& a) {
        	return point(r+a.r, i+a.i);
    	}
};


	//according to the position to do calculation. if the value is bigger than 1000, return 0; otherwise, return 1.
__device__ int calculate( int x, int y ) { 
	

    	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    	float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    	point c(-0.8, 0.156);
    	point a(jx, jy);

    	int i = 0;
    	for (i=0; i<200; i++) {
        	a = a * a + c;
        	if (a.magnitude2() > 1000)
            		return 0;
    	}

    	return 1;
}

//render each point of the image
__global__ void render( int image[DIM][DIM] ){
	int x = blockIdx.x;
	int y = blockIdx.y;
        int value = calculate( x, y );
        image[x][y] = 255 * value;
 }

int main() {
   	int image[DIM][DIM];
	int (*im_d)[DIM];
	int size = DIM*DIM*sizeof(int);
	hipMalloc((void**)&im_d,size);

	for (int y=0; y<DIM; y++) {
        	for (int x=0; x<DIM; x++) {
			image[x][y]=30;
		}
	}

	hipMemcpy(im_d,image,size, hipMemcpyHostToDevice);
	dim3 numThreads(1);
	dim3 numBlocks(DIM,DIM);
    	render<<<numBlocks,numThreads>>>(im_d);
	cerr <<  hipGetErrorString(hipGetLastError()) << endl;

	hipMemcpy(image,im_d,size, hipMemcpyDeviceToHost);
	for (int y=0; y<DIM; y++) {
        	for (int x=0; x<DIM; x++) {
			printf("%d ",image[x][y]);
		}
	printf("\n");
	}
	hipFree(im_d);
	return 0;
}
