#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 64
#define R 3

// Kernel definition
__global__ void MatMul(int A[N], int B[N])
{
	//finish matrix multiplication, each thread calculate one element of C
	int mi = threadIdx.x + blockIdx.x*blockDim.x;
	for(int j = -R; j < R+1; j++){
		if(!(mi+j < 0 || mi+j >= N)){
			B[mi] += A[mi+j];
		}
	}
}

int main() {

	int a_h[N], b_h[N];
	int (*a_d), (*b_d);

	int size = N*sizeof(int);

	//allocate the memory on the device
	hipMalloc((void**)&a_d, size);
	hipMalloc((void**)&b_d, size);

	//assign values to matrixes
	for(int i=0; i<N; i++)
	{
		a_h[i] = i;
		b_h[i] = 0;
	}



	//copy matrixes to the device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

	// launch kernel on the device by combining two-dimensional blocks with two-dimensional threads
	//define numBlocks and threadsPerBlock
	dim3 numBlocks(1);
	dim3 numThreads(N);
	MatMul<<<numBlocks, numThreads>>>(a_d, b_d);

	//copy results from the device to the host
	hipMemcpy(b_h, b_d, size, hipMemcpyDeviceToHost);



	//print the results
	for(int i=0;i<N;i++){
        		printf("%d ", b_h[i]);
    		}	

	//free the memory
	hipFree(a_d); 
	hipFree(b_d); 

	return 0;
}
