
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
#include <stdlib.h>

#define N 64
#define R 3

// Kernel definition
//__global__ void MatMul(int A[N][N], int B[N][N], int C[N][N])
//{
//	//finish matrix multiplication, each thread calculate one element of C
//	int mi = threadIdx.x + blockIdx.x*blockDim.x;
//	int mj = threadIdx.y + blockIdx.y*blockDim.y;
//	C[mi][mj] = 0;
//	for(int i = 0; i < N; i++){
//			C[mi][mj] += A[mi][i] * B[i][mj];
//	}
//}

int main() {

	int a_h[N], b_h[N];
	int (*a_d)[N], (*b_d)[N];

	int size = N*N*sizeof(int);

	////allocate the memory on the device
	//cudaMalloc((void*)&a_d, size);
	//cudaMalloc((void*)&b_d, size);

	//assign values to matrixes
	for(int i=0; i<N; i++)
	{
		a_h[i] = i;
		b_h[i] = 0;
	}


	//sequential implementation
	for(int i = 0; i < N; i++){
		for(int j = -R; j < R+1; j++){
			if(!(i+j < 0 || i+j >= N)){
				b_h[i] += a_h[i+j];
			}
		}
	}

	for(int i = 0; i < N; i++){
		printf("%d\n", b_h[i]);
	}

	//copy matrixes to the device
	//cudaMemcpy(a_d, a_h, size, cudaMemcpyHostToDevice);
	//cudaMemcpy(b_d, b_h, size, cudaMemcpyHostToDevice);
	//cudaMemcpy(c_d, c_h, size, cudaMemcpyHostToDevice);

	//// launch kernel on the device by combining two-dimensional blocks with two-dimensional threads
	////define numBlocks and threadsPerBlock
	//int nThreads = 4;
	//dim3 numBlocks(N/nThreads,N/nThreads);
	//dim3 numThreads(nThreads,nThreads);
	//MatMul<<<numBlocks, numThreads>>>(a_d, b_d, c_d);

	////copy results from the device to the host
	//cudaMemcpy(c_h, c_d, size, cudaMemcpyDeviceToHost);



	////print the results
	//for(int i=0;i<N;i++){
   	//	for(int j=0;j<N;j++){
        //		printf("%d ", c_h[i][j]);
    	//	}	
    	//	printf("\n");
	//}

	////free the memory
	//cudaFree(a_d); 
	//cudaFree(b_d); 
	//cudaFree(c_d);

	return 0;
}
