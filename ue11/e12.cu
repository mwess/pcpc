#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 32

// Kernel definition
__global__ void MatAdd(int A[N][N], int B[N][N], int C[N][N])
{
	int indexi = threadIdx.x/N;
	int indexj = threadIdx.x%N;
	if (indexi < N*N && indexj < N*N)
		C[indexi][indexj] = A[indexi][indexj] + B[indexi][indexj];
}

int main() {

	int a_h[N][N], b_h[N][N], c_h[N][N];
	int (*a_d)[N], (*b_d)[N], (*c_d)[N];

	int size = N*N*sizeof(int);

	//allocate the memory on the device
	hipMalloc((void**)&a_d, size);
	hipMalloc((void**)&b_d, size);
	hipMalloc((void**)&c_d, size);

	//assign values to matrixes
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
		{
			a_h[i][j] = j;
			b_h[i][j] = i;
		}

	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", a_h[i][j]);
    		}	
    		printf("\n\n");
	}
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", b_h[i][j]);
    		}	
    		printf("\n\n");
	}

	//copy matrixes to the device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

	// launch kernel on the device with 1 block of N*N threads
	int threadsPerBlock = N*N;
	dim3 numBlocks(1,1,1);
	MatAdd<<<numBlocks, threadsPerBlock>>>(a_d, b_d, c_d);

	//copy results from the device to the host
	hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);



	//print the results
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", c_h[i][j]);
    		}	
    		printf("\n");
	}

	//free the memory
	hipFree(a_d); 
	hipFree(b_d); 
	hipFree(c_d);

	return 0;
}
