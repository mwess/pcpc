#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 32

// Kernel definition
__global__ void MatAdd(int A[N][N], int B[N][N], int C[N][N])
{
	//how to get the element index, fill out the values of indexi and indexj
	int indexi = threadIdx.x + blockIdx.x*blockDim.x;
	int indexj = threadIdx.y + blockIdx.y*blockDim.y;
	if (indexi < N && indexj < N)
		C[indexi][indexj] = A[indexi][indexj] + B[indexi][indexj];
}

int main() {

	int a_h[N][N], b_h[N][N], c_h[N][N];
	int (*a_d)[N], (*b_d)[N], (*c_d)[N];

	int size = N*N*sizeof(int);

	//allocate the memory on the device
	hipMalloc((void**)&a_d, size);
	hipMalloc((void**)&b_d, size);
	hipMalloc((void**)&c_d, size);

	//assign values to matrixes
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
		{
			a_h[i][j] = j;
			b_h[i][j] = i;
		}

	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", a_h[i][j]);
    		}	
    		printf("\n\n");
	}
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", b_h[i][j]);
    		}	
    		printf("\n\n");
	}

	//copy matrixes to the device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

	// launch kernel on the device with 1 block of N*N threads
	int threadsPerBlock = 16;
	int nThreads = 4;
	dim3 numBlocks(N/nThreads, N/nThreads);
	dim3 numThreads(nThreads,nThreads);
	MatAdd<<<numBlocks, numThreads>>>(a_d, b_d, c_d);

	//copy results from the device to the host
	hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);



	//print the results
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", c_h[i][j]);
    		}	
    		printf("\n");
	}

	//free the memory
	hipFree(a_d); 
	hipFree(b_d); 
	hipFree(c_d);

	return 0;
}
