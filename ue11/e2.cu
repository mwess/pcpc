#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 64

// Kernel definition
__global__ void MatMul(int A[N][N], int B[N][N], int C[N][N])
{
	//finish matrix multiplication, each thread calculate one element of C
	int mi = threadIdx.x + blockIdx.x*blockDim.x;
	int mj = threadIdx.y + blockIdx.y*blockDim.y;
	C[mi][mj] = 0;
	for(int i = 0; i < N; i++){
			C[mi][mj] += A[mi][i] * B[i][mj];
	}
}

int main() {

	int a_h[N][N], b_h[N][N], c_h[N][N];
	int (*a_d)[N], (*b_d)[N], (*c_d)[N];

	int size = N*N*sizeof(int);

	//allocate the memory on the device
	hipMalloc((void**)&a_d, size);
	hipMalloc((void**)&b_d, size);
	hipMalloc((void**)&c_d, size);

	//assign values to matrixes
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
		{
			a_h[i][j] = j;
			b_h[i][j] = j;
		}

	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", a_h[i][j]);
    		}	
    		printf("\n\n");
	}
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", b_h[i][j]);
    		}	
    		printf("\n\n");
	}


	//copy matrixes to the device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

	// launch kernel on the device by combining two-dimensional blocks with two-dimensional threads
	//define numBlocks and threadsPerBlock
	int nThreads = 4;
	dim3 numBlocks(N/nThreads,N/nThreads);
	dim3 numThreads(nThreads,nThreads);
	MatMul<<<numBlocks, numThreads>>>(a_d, b_d, c_d);

	//copy results from the device to the host
	hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);



	//print the results
	for(int i=0;i<N;i++){
   		for(int j=0;j<N;j++){
        		printf("%d ", c_h[i][j]);
    		}	
    		printf("\n");
	}

	//free the memory
	hipFree(a_d); 
	hipFree(b_d); 
	hipFree(c_d);

	return 0;
}
