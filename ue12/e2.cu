
#include <hip/hip_runtime.h>
#include <stdio.h>

#define DIM 100

struct point {
	float   r;
	float   i;
	//construct function
	point( float a, float b )  {r=a; i=b;}

	float magnitude2() { return r * r + i * i; }
	
	//override the operator *
    	point operator*(const point& a) {
        	return point(r*a.r - i*a.i, i*a.r + r*a.i);
    	}

	//override the operator +
    	point operator+(const point& a) {
        	return point(r+a.r, i+a.i);
    	}
};


	//according to the position to do calculation. if the value is bigger than 1000, return 0; otherwise, return 1.
int calculate( int x, int y ) { 
	

    	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    	float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    	point c(-0.8, 0.156);
    	point a(jx, jy);

    	int i = 0;
    	for (i=0; i<200; i++) {
        	a = a * a + c;
        	if (a.magnitude2() > 1000)
            		return 0;
    	}

    	return 1;
}

//render each point of the image
void render( int image[DIM][DIM] ){
   	for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int value = calculate( x, y );
            image[x][y] = 255 * value;
        }
    	}
 }

int main() {
   	int image[DIM][DIM];

	for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
		image[x][y]=30;
	}
	}
    	render(image);

	for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
		printf("%d ",image[x][y]);
	}
	printf("\n");
	}
}
